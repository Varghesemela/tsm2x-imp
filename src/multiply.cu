#include "hip/hip_runtime.h"
/**
 * TSM2 and ISM2 Testbed and Evaluation Platform
 * by Cody Rivera, 2019-2020
 *
 * Usage - ./multiply [-d] [-i] matrixA matrixB matrixC
 * where -d signifies double precision, and -i signifies
 * ISM2
 */

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <fstream>
#include <iostream>

// #include "libwb/wb.h"
#include <hip/hip_runtime.h>
#include <rocblas/rocblas.h>
#include "hip_error.cuh"
#include "kernels.cuh"
#include "multiply.cuh"
#include "launch_rocblas.cuh"

// Testing Parameters -- Adjust as needed
#define EPS 1e-3
#define N_WARMUP 10
#define N_ROUNDS 100

/**
 * Testbed helper functions.
 */
// Based on
// https://randomascii.wordpress.com/2012/02/25/comparing-floating-point-numbers-2012-edition/
bool approxEqual(double A, double B, double maxRelDiff = EPS) {
    // Calculate the difference.
    double diff = fabs(A - B);
    A = fabs(A);
    B = fabs(B);
    // Find the largest
    double largest = (B > A) ? B : A;

    if (diff <= largest * maxRelDiff) return true;
    return false;
}

template <typename FloatType>
bool matrixCompare(const FloatType* A, const FloatType* B, unsigned int m,
                   unsigned int n, unsigned int& iFail, unsigned int& jFail) {
    FloatType aVal, bVal;
    bool b = true;
    // Cache-friendly comparison pattern
    for (unsigned int j = 0; j < n && b; j++) {
        for (unsigned int i = 0; i < m && b; i++) {
            aVal = A[i + (j * m)];
            bVal = B[i + (j * m)];
            if (!approxEqual(aVal, bVal, EPS)) {
                iFail = i;
                jFail = j;
                b = false;
            }
        }
    }
    return b;
}

template <typename FloatType>
void reportTestSuccess(const char* testName, double GFLOPs) {
    printf("%s succeeded: %g GFLOPs, %g GFLOPs acc. for transfers\n", testName,
           GFLOPs);
}

template <typename FloatType>
void reportTestFailure(const char* testName, const FloatType* orig,
                       const FloatType* cand, unsigned int leadDim,
                       unsigned int iFail, unsigned int jFail) {
    double oVal = (double)orig[iFail + (jFail * leadDim)];
    double cVal = (double)cand[iFail + (jFail * leadDim)];
    fprintf(stderr,
            "%s failed: Original[%u, %u] = %.6f != Candidate[%u, %u] = %.6f\n",
            testName, iFail, jFail, oVal, iFail, jFail, cVal);
}

template <typename FloatType>
double getGFLOPs(double time, unsigned int m, unsigned int n, unsigned int k) {
    double instCount = ((double)m * (double)n * (double)k) / 1e9;
    double timeSeconds = time / 1000;
    return instCount / timeSeconds;
}

/**
 * Kernel launch wrapper. Runs both rocBLAS and TSM2/ISM2, for evaluation
 * purposes.
 */
template <typename FloatType>
bool runKernels(const FloatType* A, const FloatType* B, FloatType* C,
                const unsigned int m, const unsigned int n,
                const unsigned int k, const bool runIsm2) {
    // Candidate for C -- Used by GPU kernels
    FloatType* candC;
    // Device memory
    FloatType *devA, *devB, *devC;
    // Events used for timing
    hipEvent_t start, end, startTotal, endTotal;
    float time, timeTotal;

    printf("Multiplying matrix A[%u, %u] by matrix B[%u, %u]\n\n", m, k, k, n);

    // Change test name depending on runIsm2
    const char* testName = "TSM2 Kernel Test";
    if (runIsm2) {
        testName = "ISM2 Kernel Test";
    }

    // Allocates new memory
    candC = (FloatType*)malloc(m * n * sizeof(FloatType));
    if (candC == NULL) {
        fprintf(stderr, "Not enough memory\n");
        return false;
    }

    hipMalloc((FloatType**)&devA, m * k * sizeof(FloatType));
    hipMalloc((FloatType**)&devB, k * n * sizeof(FloatType));
    hipMalloc((FloatType**)&devC, m * n * sizeof(FloatType));

    // Inits hip events
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventCreate(&startTotal);
    hipEventCreate(&endTotal);

    
    // Runs rocBLAS call
    rocblas_handle handle;
    rocblas_create_handle(&handle);

    FloatType one = 1;
    FloatType zero = 0;

    
    hipEventRecord(startTotal);

    // hip Memory Copy
    
    hipMemcpy(devA, A, m * k * sizeof(FloatType), hipMemcpyHostToDevice);
    hipMemcpy(devB, B, k * n * sizeof(FloatType), hipMemcpyHostToDevice);

    for (int i = 0; i < N_WARMUP; ++i) {
        launchRocblas<FloatType>(handle, one, zero, devA, devB, devC, m, n, k);
    }
    
    hipEventRecord(start);
    for (int i = 0; i < N_ROUNDS; ++i) {
        launchRocblas<FloatType>(handle, one, zero, devA, devB, devC, m, n, k);
    }
    hipEventRecord(end);

    // Copies result back
    
    hipMemcpy(C, devC, m * n * sizeof(FloatType), hipMemcpyDeviceToHost);

    hipEventRecord(endTotal);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, end);
    hipEventElapsedTime(&timeTotal, startTotal, endTotal);
    time /= N_ROUNDS;
    timeTotal /= N_ROUNDS;

    reportTestSuccess<FloatType>("rocBLAS Test", getGFLOPs<FloatType>(time, m, n, k));

    rocblas_destroy_handle(handle);
    

    // Runs kernel
    // Failure flag
    bool status;
    // Failure indices
    unsigned int iFail, jFail;

    // Clear result matrix
    hipMemset(devC, 0, m * n * sizeof(FloatType));
    hipEventRecord(startTotal);

    // hip Memory Copy
    
    hipMemcpy(devA, A, m * k * sizeof(FloatType), hipMemcpyHostToDevice);

    hipMemcpy(devB, B, k * n * sizeof(FloatType), hipMemcpyHostToDevice);

    for (int i = 0; i < N_WARMUP; ++i) {
        hipMemset(devC, 0, m * n * sizeof(FloatType));
        if (runIsm2) {
            launchKernelIsm2(devA, devB, devC, m, n, k);
        } else {
            launchKernelTsm2(devA, devB, devC, m, n, k);
        }
    }
    
    hipEventRecord(start);
    for (int i = 0; i < N_ROUNDS; ++i) {
        hipMemset(devC, 0, m * n * sizeof(FloatType));
        if (runIsm2) {
            launchKernelIsm2(devA, devB, devC, m, n, k);
        } else {
            launchKernelTsm2(devA, devB, devC, m, n, k);
        }
    }
    hipGetLastError();
    hipEventRecord(end);

    // Copies result back
    hipMemcpy(candC, devC, m * n * sizeof(FloatType),
                          hipMemcpyDeviceToHost);

    hipEventRecord(endTotal);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time, start, end);
    hipEventElapsedTime(&timeTotal, startTotal, endTotal);
    time /= N_ROUNDS;
    timeTotal /= N_ROUNDS;
    
    status = matrixCompare<FloatType>(C, candC, m, n, iFail, jFail);
    if (status) {
        reportTestSuccess<FloatType>(testName,
                                  getGFLOPs<FloatType>(time, m, n, k));
    } else {
        reportTestFailure<FloatType>(testName, C, candC, m, iFail, jFail);
    }

    hipEventDestroy(start);
    hipEventDestroy(end);
    hipEventDestroy(startTotal);
    hipEventDestroy(endTotal);
    free(candC);
    hipFree(devA);
    hipFree(devB);
    hipFree(devC);

    return true;
}

/**
 * Runs testbed on specified input files. Handles file IO.
 */
template <typename FloatType>
bool runMatmul(std::istream& fileA, std::istream& fileB, std::ostream& outFile,
               bool runIsm2) {
    FloatType *A, *B, *C;
    int m, n, k, kCand;

    // Reads Matrix Sizes
    fileA.read((char*)&m, sizeof(unsigned int));
    fileA.read((char*)&k, sizeof(unsigned int));
    fileB.read((char*)&kCand, sizeof(unsigned int));
    fileB.read((char*)&n, sizeof(unsigned int));

    if (k != kCand) {
        fprintf(stderr,
                "Matrix multiplication is undefined where A's"
                "column count is not equal\n to B's row count\n\n"
                "Matrix A (%u x %u) and Matrix B (%u x %u)\n",
                m, k, kCand, n);
        return false;
    }

    // Mallocs Matrices on CPU
    A = (FloatType*)malloc((size_t)m * k * sizeof(FloatType));
    B = (FloatType*)malloc((size_t)k * n * sizeof(FloatType));
    C = (FloatType*)malloc((size_t)m * n * sizeof(FloatType));

    if (A == NULL || B == NULL || C == NULL) {
        fprintf(stderr, "Not enough memory\n");
        return false;
    }

    // Loads Data to Matrix A and B
    fileA.read((char*)A, (size_t)m * k * sizeof(FloatType));
    fileB.read((char*)B, (size_t)k * n * sizeof(FloatType));

    // Calls hip
    bool status = runKernels<FloatType>(A, B, C, m, n, k, runIsm2);
    if (!status) {
        free(A);
        free(B);
        free(C);
        return false;
    }

    // Writes output matrix
    outFile.write((const char*)&m, sizeof(unsigned int));
    outFile.write((const char*)&n, sizeof(unsigned int));
    outFile.write((const char*)C, (size_t)m * n * sizeof(FloatType));

    free(A);
    free(B);
    free(C);
    return true;
}

/**
 * Entry point
 */
int main(int argc, char** argv) {
    int fileArg[3];
    int nFiles = 0;
    bool isDouble = false;
    bool runIsm2 = false;
    for (int i = 1; i < argc; i++) {
        if (strcmp(argv[i], "-d") == 0) {
            isDouble = true;
        } else if (strcmp(argv[i], "-i") == 0) {
            runIsm2 = true;
        } else {
            if (nFiles < 3) {
                fileArg[nFiles] = i;
            }
            nFiles++;
        }
    }
    if (nFiles != 3) {
        fprintf(stderr, "Usage: %s [-d] [-i] matrixA matrixB matrixC\n",
                argv[0]);
        return 1;
    }

    std::ifstream fileA(argv[fileArg[0]], std::ios::binary),
        fileB(argv[fileArg[1]], std::ios::binary);
    std::ofstream outFile(argv[fileArg[2]], std::ios::binary);
    if (!fileA) {
        fprintf(stderr, "Cannot open %s for reading\n", argv[fileArg[0]]);
        return 1;
    }
    if (!fileB) {
        fprintf(stderr, "Cannot open %s for reading\n", argv[fileArg[1]]);
        return 1;
    }
    if (!outFile) {
        fprintf(stderr, "Cannot open %s for writing\n", argv[fileArg[2]]);
        return 1;
    }
    // Runs matmul
    bool status = false;
    if (isDouble) {
        status = runMatmul<double>(fileA, fileB, outFile, runIsm2);
    } else {
        status = runMatmul<float>(fileA, fileB, outFile, runIsm2);
    }
    fileA.close();
    fileB.close();
    outFile.close();
    if (status) {
        return 0;
    } else {
        return 1;
    }
}

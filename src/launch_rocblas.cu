/**
 * rocblas wrapper for different matrix types
 * by Cody Rivera, 2019-2020
 */

#include "rocblas.h"
#include "launch_rocblas.cuh"


// float specialization
template <>
rocblasStatus_t launchrocblas(rocblasHandle_t handle, float& one, float& zero,
                            const float* devA, const float* devB, float* devC,
                            const unsigned int m, const unsigned int n, 
                            const unsigned int k) {
    return rocblasGemmEx(handle, rocblas_OP_N, rocblas_OP_N, m, n, k, &one,
                        devA, HIP_R_32F, m, devB, HIP_R_32F, k, &zero,
                        devC, HIP_R_32F, m, HIP_R_32F,
                        rocblas_GEMM_DEFAULT);
}

// double specialization
template <>
rocblasStatus_t launchrocblas(rocblasHandle_t handle, double& one, double& zero,
                            const double* devA, const double* devB, double* devC,
                            const unsigned int m, const unsigned int n, 
                            const unsigned int k) {
    return rocblasGemmEx(handle, rocblas_OP_N, rocblas_OP_N, m, n, k, &one,
                        devA, HIP_R_64F, m, devB, HIP_R_64F, k, &zero,
                        devC, HIP_R_64F, m, HIP_R_64F,
                        rocblas_GEMM_DEFAULT);
}
 